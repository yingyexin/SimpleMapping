#include "hip/hip_runtime.h"
// Copyright 2018 Emanuele Palazzolo (emanuele.palazzolo@uni-bonn.de), Cyrill Stachniss, University of Bonn
#include "rgbd_image.h"
#include "utils.h"

namespace refusion {

    RgbdImage::~RgbdImage() {
        hipDeviceSynchronize();
        hipFree(rgb_);
        hipFree(depth_);
    }

    void RgbdImage::Init(const RgbdSensor &sensor) {
        sensor_ = sensor;
        gpuErrchk(hipMallocManaged(&rgb_, sizeof(uchar3) * sensor_.rows * sensor.cols));
        gpuErrchk(hipMallocManaged(&depth_, sizeof(float) * sensor_.rows * sensor.cols));
        gpuErrchk(hipDeviceSynchronize());
    }

    __host__ __device__ inline float3 RgbdImage::GetPoint3d(int u, int v) const {
        float3 point;
        point.z = depth_[v * sensor_.cols + u];
        point.x = (static_cast<float>(u) - sensor_.cx) * point.z / sensor_.fx;
        point.y = (static_cast<float>(v) - sensor_.cy) * point.z / sensor_.fy;
        return point;
    }

    __host__ __device__ inline float3 RgbdImage::GetPoint3d(int i) const {
        int v = i / sensor_.cols;
        int u = i - sensor_.rows * v;
        return GetPoint3d(u, v);
    }

}  // namespace refusion
